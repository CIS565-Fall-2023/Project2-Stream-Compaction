#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include "common.h"
#include "efficient.h"

#define blockSize 128

#define TIME_COMPACT 1

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
 
        __global__ void kernUpSweep(int n, int d, int* odata) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (2 << d);
 
            if (index >= n) return;

            odata[index + (1 << (d + 1)) - 1] += odata[index + (1 << d) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* odata) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (2 << d);

            if (index >= n) return;

            // preserve the left child value
            int temp = odata[index + (1 << d) - 1];
            // left child copies the parent value
            odata[index + (1 << d) - 1] = odata[index + (1 << (d + 1)) - 1];
            // right child addes the parent value and the preserved left child value
            odata[index + (1 << (d + 1)) - 1] += temp;
        }

        // apply shared memory to scan each block
        __global__ void kernBlockScan(int n, int* odata, const int* idata, int* blockSums) {
            extern __shared__ int temp[];

            int thid = threadIdx.x;
            int index = blockIdx.x * blockDim.x * 2 + thid;

            // Load input into shared memory with boundary checks
            temp[2 * thid] = (2 * index < n) ? idata[2 * index] : 0;
            temp[2 * thid + 1] = (2 * index + 1 < n) ? idata[2 * index + 1] : 0;

            int offset = 1;

            // Up-sweep (reduce) phase
            for (int d = blockDim.x; d > 0; d >>= 1) {
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }

            // Clear last element
            if (thid == 0) {
                temp[2 * blockDim.x - 1] = 0;
            }

            // Down-sweep phase
            for (int d = 1; d < 2 * blockDim.x; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();

            // Write results to device memory with boundary checks
            if (2 * index < n) {
                odata[2 * index] = temp[2 * thid];
                if (2 * index + 1 < n) {
                    odata[2 * index + 1] = temp[2 * thid + 1];
                }
            }

            // Save block sum
            if (thid == 0) {
                blockSums[blockIdx.x] = temp[2 * blockDim.x - 2] + temp[2 * blockDim.x - 1];
            }
        }


        __global__ void kernAddScannedBlockSums(int n, int* odata, const int* blockSums) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) return;

            if (blockIdx.x > 0) {
                odata[index] += blockSums[blockIdx.x - 1];
            }
        }

        /**
         * Performs prefix-sum (aks scan) on idata using the shared memory, storing the result into odata
         */
        void scanShared(int n, int* odata, const int* idata) {
            int* dev_in, * dev_out, * dev_blockSums;
            
            const int log2ceil = ilog2ceil(n);
            const long int fullSize = 1 << log2ceil;

            int gridSize = (fullSize + blockSize - 1) / blockSize;
            // printf("gridSize: %d\n", gridSize);

            // allocate gpu memory
            hipMalloc((void**)&dev_in, fullSize * sizeof(int));
            hipMemset(dev_in, 0, fullSize * sizeof(int));
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            hipMalloc((void**)&dev_out, n * sizeof(int));

            hipMalloc((void**)&dev_blockSums, gridSize * sizeof(int));
            checkCUDAErrorFn("malloc dev_blockSums failed!");

            timer().startGpuTimer();
            kernBlockScan << <gridSize, blockSize, 2 * blockSize * sizeof(int) >> > (fullSize, dev_out, dev_in, dev_blockSums);

            int* blockSums = new int[gridSize];
            hipMemcpy(blockSums, dev_blockSums, gridSize * sizeof(int), hipMemcpyDeviceToHost);

            printf("blockSums\n");
            for (int i = 0; i < gridSize; ++i) {
                printf("%d ", blockSums[i]);
            }
            printf("\n");

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            printf("odata\n");
            for (int i = 0; i < n; ++i) {
                printf("%d ", odata[i]);
            }
            printf("\n");

            // Assuming gridSize is small enough for a single block to handle
            kernBlockScan << <1, gridSize / 2, gridSize * sizeof(int) >> > (gridSize, dev_blockSums, dev_blockSums, nullptr);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_in);
            hipFree(dev_out);
            hipFree(dev_blockSums);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_out;

            const int log2ceil = ilog2ceil(n);
            const long int fullSize = 1 << log2ceil;

            hipMalloc((void**)&dev_out, fullSize * sizeof(int));
            hipMemset(dev_out, 0, fullSize * sizeof(int));
            hipMemcpy(dev_out, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            // up sweep 
            for (int d = 0; d <= log2ceil - 1; ++d) {
                // Adjust the grid size based on the depth of the sweep
                int gridSize = (fullSize / (2 << d) + blockSize - 1) / blockSize;
                kernUpSweep << <gridSize, blockSize >> > (fullSize, d, dev_out);
                checkCUDAErrorFn("up sweep failed!");
            }

            // set the last value to 0
            hipMemset(dev_out + fullSize - 1, 0, sizeof(int));
            checkCUDAErrorWithLine("set the last value to zero failed!");

            // down sweep
            for (int d = log2ceil - 1; d >= 0; --d) { 
                // Adjust the grid size based on the depth of the sweep
                int gridSize = (fullSize / (2 << d) + blockSize - 1) / blockSize;
                kernDownSweep << <gridSize, blockSize >> > (fullSize, d, dev_out);
                checkCUDAErrorFn("down sweep failed");
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_out);
        }


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int* dev_in, * dev_out, * dev_bools, * dev_scan;

            int boolLastVal, scanLastVal;

            int gridSize = (n + blockSize - 1) / blockSize;

            hipMalloc((void**)&dev_in, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_in failed!");
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("copy idata to dev_in failed!");

            hipMalloc((void**)&dev_out, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_out failed!");

            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_bools failed!");

#if TIME_COMPACT
            const int log2ceil = ilog2ceil(n);
            const long int fullSize = 1 << log2ceil;

            hipMalloc((void**)&dev_scan, fullSize * sizeof(int));
            checkCUDAErrorFn("malloc dev_scan failed!");
            hipMemset(dev_scan, 0, n * sizeof(int));
#else
            hipMalloc((void**)&dev_scan, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_scan failed!");
#endif

#if TIME_COMPACT
            timer().startGpuTimer();
#endif
            // map the bool array
            StreamCompaction::Common::kernMapToBoolean << <gridSize, blockSize >> > (n, dev_bools, dev_in);
            checkCUDAErrorFn("map bool array failed!");

            
#if TIME_COMPACT
            // scan the bool array
            hipMemcpy(dev_scan, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            // up sweep
            for (int d = 0; d <= log2ceil - 1; ++d) {
                int dynamicGridSize = (fullSize / (2 << d) + blockSize - 1) / blockSize;
                kernUpSweep << <dynamicGridSize, blockSize >> > (fullSize, d, dev_scan);
                checkCUDAErrorFn("up sweep failed!");
            }

            // set the last value to 0
            hipMemset(dev_scan + fullSize - 1, 0, sizeof(int));
            
            // down sweep
            for (int d = log2ceil - 1; d >= 0; --d) {
                int dynamicGridSize = (fullSize / (2 << d) + blockSize - 1) / blockSize;
                kernDownSweep << <dynamicGridSize, blockSize >> > (fullSize, d, dev_scan);
                checkCUDAErrorFn("down sweep failed");
            }
#else 
            scan(n, dev_scan, dev_bools);
#endif
            // scatter
            StreamCompaction::Common::kernScatter << <gridSize, blockSize >> > (n, dev_out, dev_in, dev_bools, dev_scan);
            checkCUDAErrorFn("scatter failed!");
#if TIME_COMPACT
            timer().endGpuTimer();
#endif
            // store the last value of the bool array
            hipMemcpy(&boolLastVal, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy last bool value to host failed!");

            // store the last value of the scan results
            hipMemcpy(&scanLastVal, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy last bool value to host failed!");

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy dev_out to odata failed!");

            // free memory
            hipFree(dev_in);
            hipFree(dev_out);
            hipFree(dev_bools);
            hipFree(dev_scan);

            return scanLastVal + boolLastVal;
        }
    }
}
