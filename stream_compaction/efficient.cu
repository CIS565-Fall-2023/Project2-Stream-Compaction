#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 256
#define OPTIMIZED 1
#define TIMESCAN 1

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* data) 
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            int powerd = 1 << d;
            int powerdp1 = 1 << (d + 1);

            if (k >= n || k % powerdp1) return;

            data[k + powerdp1 - 1] += data[k + powerd - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* data) 
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;           

            int powerd = 1 << d;
            int powerdp1 = 1 << (d + 1);

            if (k >= n || k % powerdp1 || k + powerdp1 - 1 >= n) return;

            int t = data[k + powerd - 1];
            data[k + powerd - 1] = data[k + powerdp1 - 1];
            data[k + powerdp1 - 1] += t;
        }

        __global__ void kernOptUpSweep(int n, int d, int offset, int* data)
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (k >= n || k >= d) return;

            int i = offset * (2 * k + 1) - 1;
            int j = offset * (2 * k + 2) - 1;

            data[j] += data[i];
        }

        __global__ void kernOptDownSweep(int n, int d, int offset, int* data)
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (k >= n || k >= d) return;

            int i = offset * (2 * k + 1) - 1;
            int j = offset * (2 * k + 2) - 1;

            int t = data[i];
            data[i] = data[j];
            data[j] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
        {
            int N = 1 << ilog2ceil(n);

            int* dev_data;

            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMemset(dev_data, 0, N * sizeof(int));
            checkCUDAErrorFn("hipMemset dev_data to 0 failed!");
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy idata to dev_data failed!");

            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
#if OPTIMIZED
            int offset = 1;
#endif

#if TIMESCAN
            timer().startGpuTimer();
#endif
            // TODO
#if OPTIMIZED
            for (int d = N >> 1; d > 0; d >>= 1) 
            {
                fullBlocksPerGrid = dim3((d + blockSize - 1) / blockSize);
                kernOptUpSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, offset, dev_data);
                offset <<= 1;
            }

#else
            for (int d = 0; d < ilog2ceil(N); d++)
            {
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }
#endif

            hipMemset(dev_data + N - 1, 0, sizeof(int));
#if OPTIMIZED
            for (int d = 1; d < N; d <<= 1)
            {
                offset >>= 1;
                fullBlocksPerGrid = dim3((d + blockSize - 1) / blockSize);
                kernOptDownSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, offset, dev_data);
            }
#else
            for (int d = ilog2ceil(N) - 1; d >= 0; d--) 
            {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }
#endif

#if TIMESCAN
            timer().endGpuTimer();
#endif

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy dev_data to odata failed!");

            hipFree(dev_data);
            checkCUDAErrorFn("hipFree dev_data failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
        {
            int N = 1 << ilog2ceil(n);

            int* dev_bools;
            int* dev_data;
            int* dev_idata;
            int* dev_odata;

            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_odata failed!");

            hipMemset(dev_data, 0, N * sizeof(int));
            checkCUDAErrorFn("hipMemset dev_data failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("cudaMempcy idata to dev_idata failed!");

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
            StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_bools, dev_idata);

            hipMemcpy(dev_data, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAErrorFn("cudaMempcy dev_bools to dev_data failed!");

            for (int d = 0; d < ilog2ceil(N); d++)
            {
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }

            hipMemset(dev_data + N - 1, 0, sizeof(int));
            for (int d = ilog2ceil(N) - 1; d >= 0; d--)
            {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }

            StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_data);

            timer().endGpuTimer();

            int count = 0;
            hipMemcpy(&count, dev_data + N - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("cudaMempcy count failed!");
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("cudaMempcy dev_odata to odata failed!");

            hipFree(dev_bools);
            checkCUDAErrorFn("hipFree dev_bools failed!");
            hipFree(dev_data);
            checkCUDAErrorFn("hipFree dev_data failed!");
            hipFree(dev_idata);
            checkCUDAErrorFn("hipFree dev_idata failed!");
            hipFree(dev_odata);
            checkCUDAErrorFn("hipFree dev_odata failed!");

            return count;
        }

        __global__ void kernComputeEArray(int n, int bit, int* edata, const int* idata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            edata[index] = !((idata[index] >> bit) & 1);
        }

        __global__ void kernComputeTArray(int n, int totalFalses, int* tdata, const int* fdata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            tdata[index] = index - fdata[index] + totalFalses;
        }

        __global__ void kernComputeDArray(int n, int* ddata, const int* edata, const int* tdata, const int* fdata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            ddata[index] = edata[index] ? fdata[index] : tdata[index];
        }

        __global__ void kernScatter(int n, int* ddata, int* odata, int* idata) 
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            odata[ddata[index]] = idata[index];
        }

        void radixSort(int n, int* odata, const int* idata)
        {
            int* dev_edata;
            int* dev_fdata;
            int* dev_tdata;
            int* dev_ddata;

            int* dev_idata;
            int* dev_odata;

            hipMalloc((void**)&dev_edata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_edata failed!");
            hipMalloc((void**)&dev_fdata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_fdata failed!");
            hipMalloc((void**)&dev_tdata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_tdata failed!");
            hipMalloc((void**)&dev_ddata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_ddata failed!");

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("cudaMempcy idata to dev_idata failed!");

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int bnum = ilog2ceil(*(std::max_element(idata, idata + n)));
            
            timer().startGpuTimer();
            for (int d = 0; d < bnum; d++) 
            {
                // Step1: Compute e array
                kernComputeEArray<<<fullBlocksPerGrid, blockSize>>>(n, d, dev_edata, dev_idata);
                // Step2: Scan e
                scan(n, dev_fdata, dev_edata);
                // Step3: Compute totalFalse
                int e_last;
                int f_last;
                hipMemcpy(&e_last, dev_edata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&f_last, dev_fdata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                int totalFalses = e_last + f_last;
                // Step4: Compute t
                kernComputeTArray<<<fullBlocksPerGrid, blockSize>>>(n, totalFalses, dev_tdata, dev_fdata);
                // Step5: scatter
                kernComputeDArray<<<fullBlocksPerGrid, blockSize>>>(n, dev_ddata, dev_edata, dev_tdata, dev_fdata);
                kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_ddata, dev_odata, dev_idata);
                hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
                checkCUDAErrorFn("cudaMempcy dev_odata to dev_idata failed!");
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("cudaMempcy dev_odata to odata failed!");

            hipFree(dev_edata);
            checkCUDAErrorFn("hipFree dev_edata failed!");
            hipFree(dev_fdata);
            checkCUDAErrorFn("hipFree dev_fdata failed!");
            hipFree(dev_tdata);
            checkCUDAErrorFn("hipFree dev_tdata failed!");
            hipFree(dev_ddata);
            checkCUDAErrorFn("hipFree dev_ddata failed!");
            hipFree(dev_idata);
            checkCUDAErrorFn("hipFree dev_idata failed!");
            hipFree(dev_odata);
            checkCUDAErrorFn("hipFree dev_odata failed!");
        }
    }
}
