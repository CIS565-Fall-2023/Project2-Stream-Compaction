#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

static int blockSize = 256;

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        __global__ void kernelUpSweep(int n, int offset, int* i_odata) {
            //int id = threadIdx.x + blockDim.x * blockIdx.x;
            //if (id >= n || (id + 1) % (offset * 2) != 0)return;
            int threadId = threadIdx.x + blockDim.x * blockIdx.x + 1;
            int id = offset * 2 * threadId - 1;
            if (id >= n)return;
            i_odata[id] = i_odata[id] + i_odata[id - offset];
        }

        __global__ void kernelDownSweep(int n, int offset, int* i_odata) {
            //int id = threadIdx.x + blockDim.x * blockIdx.x;
            //if (id >= n || (id + 1) % (offset * 2)!=0)return;
            int threadId = threadIdx.x + blockDim.x * blockIdx.x + 1;
            int id = offset * 2 * threadId - 1;
            if (id >= n)return;
            //change 2
            int prevIdx = id - offset;
            int prevNum = i_odata[prevIdx];
            i_odata[prevIdx] = i_odata[id];
            i_odata[id] += prevNum;
        }

        void devScan(int* dev_data, int layerCnt, int blockSize) {
            int N = 1 << layerCnt;
            int offset = 1;
            int needN = N;
            for (int i = 0;i < layerCnt;++i) {
                dim3 blockPerGrid((needN + blockSize - 1) / blockSize);
                kernelUpSweep << <blockPerGrid, blockSize >> > (N, offset, dev_data);
                offset *= 2;
                needN /= 2;
            }
            hipMemset(dev_data + offset - 1,0,sizeof(int));
            for (int i = 0;i < layerCnt;++i) {
                offset /= 2;
                dim3 blockPerGrid((needN + blockSize - 1) / blockSize);
                kernelDownSweep << <blockPerGrid, blockSize >> > (N, offset, dev_data);
                needN *= 2;
            }
        }

        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int* dev_data;
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            devScan(dev_data, layerCnt, blockSize);

            //exclusive scan
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_data);
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startCpuTimer();
            // TODO
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;
            hipMalloc((void**)&dev_idata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_indices, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            
            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
            StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid,blockSize>>>(N, dev_bools, dev_idata);
            
            hipMemcpy(dev_indices, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
            devScan(dev_indices, layerCnt,blockSize);
            
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (N, dev_odata, dev_idata, dev_bools, dev_indices);
            
            //read GPU
            int ans = 0;
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipMemcpy(&ans, dev_indices + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            int lastBool = 0;
            hipMemcpy(&lastBool, dev_bools + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            ans += lastBool;
            
            //free GPU
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_odata);
            hipFree(dev_indices);
            timer().endCpuTimer();
            return ans;
        }

        __global__ void kernMapToBoolean(int n, int* bools, const int* idata, int mask, bool recordZero) {
            // TODO
            int id = blockDim.x * blockIdx.x + threadIdx.x;
            if (id >= n)return;
            bools[id] = (idata[id] & mask) == 0 ? recordZero : (!recordZero);
        }

        __global__ void kernSortScatter(int n, int* odata,
            const int* idata, const int* isOneBools, 
            const int* indices_0,const int* indices_1,int zeroCnt) {
            // TODO
            int id = blockDim.x * blockIdx.x + threadIdx.x;
            if (id >= n)return;
            if (isOneBools[id] == 1) {
                int idx = indices_1[id] + zeroCnt;
                odata[idx] = idata[id];
            }
            else {
                int idx = indices_0[id];
                odata[idx] = idata[id];
            }
        }

        void sort(int n, int* odata, const int* idata) {
            timer().startCpuTimer();
            // TODO
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices_1;
            int* dev_indices_0;
            hipMalloc((void**)&dev_idata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_indices_1, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_indices_0, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(dev_idata + n, INT_MAX, (N - n)*sizeof(int));//to make non-power-of-two right

            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
            int mask = 1;
            for (int i = 0;i < 32;++i) {
                //map 0
                kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (N, dev_bools, dev_idata,mask,true);
                hipMemcpy(dev_indices_0, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
                devScan(dev_indices_0, layerCnt, blockSize);
                int zeroCnt = 0;
                int lastBool = 0;
                hipMemcpy(&zeroCnt, dev_indices_0 + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastBool, dev_bools + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
                zeroCnt += lastBool;

                //map1
                kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (N, dev_bools, dev_idata, mask, false);
                hipMemcpy(dev_indices_1, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
                devScan(dev_indices_1, layerCnt, blockSize);
                kernSortScatter << <fullBlocksPerGrid, blockSize >> > (N, 
                    dev_odata, dev_idata, dev_bools, dev_indices_0, dev_indices_1,zeroCnt);
                mask <<= 1;
                std::swap(dev_odata, dev_idata);
            }

            //read GPU
            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);

            //free GPU
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_odata);
            hipFree(dev_indices_0);
            hipFree(dev_indices_1);
            timer().endCpuTimer();
        }
    }
}
