#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

#define BLOCK_SIZE 128

namespace StreamCompaction {
  namespace Efficient {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    // padded_n must be a power of 2
    __global__ void up_sweep(int* data, int d, int num_thds) {
      int thd_idx = threadIdx.x + blockIdx.x * blockDim.x;
      if (thd_idx >= num_thds) {
        // block not full, terminate threads early
        // number of early terminated threads < BLOCK_SIZE always
        return;
      }
      // index of the element in the array that will be updated
      int arr_idx = ((thd_idx + 1) << (d + 1)) - 1;
      // index of the element whose value will be added to data[arr_idx]
      int add_idx = arr_idx - (1 << d);
      // update element
      data[arr_idx] += data[add_idx];
    }

    __global__ void down_sweep(int* data, int d, int num_thds) {
      int thd_idx = threadIdx.x + blockIdx.x * blockDim.x;
      if (thd_idx >= num_thds) {
        // block not full, terminate threads early
        // number of early terminated threads < BLOCK_SIZE always
        return;
      }
      // index of the left cell, which will inherit value from r_idx
      int l_idx = (thd_idx << (d + 1)) + (1 << d) - 1;
      int r_idx = l_idx + (1 << d);
      int tmp = data[l_idx];
      data[l_idx] = data[r_idx];
      data[r_idx] += tmp;
    }

    __global__ void nullify_last_elem(int padded_n, int* data) {
      data[padded_n - 1] = 0;
    }

    /**
      * Performs prefix-sum (aka scan) on idata, storing the result into odata.
      */
    void scan(int n, int *odata, const int *idata) {
      int layer = ilog2ceil(n);
      int padded_n = 1 << layer;
      int num_thds = padded_n;
      int *dev_buffer;
      hipMalloc((void**)&dev_buffer, padded_n * sizeof(int));
      checkCUDAError("failed to hipMalloc buffer");
      hipMemcpy(dev_buffer, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("failed to copy idata to buffer");
      timer().startGpuTimer();
      // TODO
      for (int d = 0; d < layer; d++) {
        // update #threads needed
        num_thds >>= 1;
        dim3 gridDim((num_thds + BLOCK_SIZE - 1) / BLOCK_SIZE);
        up_sweep<<<gridDim, BLOCK_SIZE>>>(dev_buffer, d, num_thds);
      }
      nullify_last_elem<<<1, 1>>>(padded_n, dev_buffer);
      for (int d = layer - 1; d >= 0; d--) {
        dim3 gridDim((num_thds + BLOCK_SIZE - 1) / BLOCK_SIZE);
        down_sweep<<<gridDim, BLOCK_SIZE>>>(dev_buffer, d, num_thds);
        num_thds <<= 1;
      }
      timer().endGpuTimer();
      hipMemcpy(odata, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("failed to copy buffer to odata");
      hipFree(dev_buffer);
      checkCUDAError("failed to free dev_buffer");
    }

    /**
      * Performs stream compaction on idata, storing the result into odata.
      * All zeroes are discarded.
      *
      * @param n      The number of elements in idata.
      * @param odata  The array into which to store elements.
      * @param idata  The array of elements to compact.
      * @returns      The number of elements remaining after compaction.
      */
    int compact(int n, int *odata, const int *idata) {
      timer().startGpuTimer();
      // TODO
      timer().endGpuTimer();
      return -1;
    }
  }
}
