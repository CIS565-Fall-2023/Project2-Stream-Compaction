#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""
#include <iostream>

#define BLOCK_SIZE 16

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void efficientScanUpSweep(int n, int nThread, int d, int *data) {
            int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= nThread) return;
            int currIdx = ((idx + 1) << (d + 1)) - 1;
            int prevIdx = currIdx - (1 << d);
            data[currIdx] += data[prevIdx];
        }

        __global__ void efficientScanDownSweep(int n, int nThread, int d, int *data) {
            int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= nThread) return;
            int currIdx = (n >> d) * (idx + 1) - 1;
            int prevIdx = currIdx - (n >> (d + 1));
            int temp = data[currIdx];
            data[currIdx] += data[prevIdx];
            data[prevIdx] = temp;
        }

        __global__ void replaceWithZero(int n, int nThread, int* data) {
            int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= nThread) return;
            data[n - 1] = 0;
        }

        void efficientScanUpDownSweep(int n, int newN, int* dev_idata) {
            dim3 numBlocks;
            int nThread = newN;
            // up sweep
            for (int d = 0; d < ilog2ceil(n); d++) {
                nThread = newN >> (d + 1);
                numBlocks = (nThread + BLOCK_SIZE - 1) / BLOCK_SIZE;
                efficientScanUpSweep <<<numBlocks, BLOCK_SIZE >>> (newN, nThread, d, dev_idata);
            }
            // replace the last number of the array with 0.
            replaceWithZero <<<1, 1 >>> (newN, 1, dev_idata);
            // down sweep
            for (int d = 0; d < ilog2ceil(n); d++) {
                nThread = 1 << d;
                numBlocks = (nThread + BLOCK_SIZE - 1) / BLOCK_SIZE;
                efficientScanDownSweep <<<numBlocks, BLOCK_SIZE >>> (newN, nThread, d, dev_idata);
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int newN = 1 << ilog2ceil(n);
            hipMalloc((void**)&dev_idata, sizeof(int) * newN);
            checkCUDAError("Efficient scan: hipMalloc failed (dev_idata)");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            timer().startGpuTimer();
            //// TODO
            efficientScanUpDownSweep(n, newN, dev_idata);

            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int newN = 1 << ilog2ceil(n);
            int* dev_idata;
            hipMalloc((void**)&dev_idata, sizeof(int) * newN);
            checkCUDAError("Efficient scan: hipMalloc failed (dev_idata)");
            int* dev_bools;
            hipMalloc((void**)&dev_bools, sizeof(int) * newN);
            checkCUDAError("Efficient scan: hipMalloc failed (dev_bools)");
            int* dev_indices;
            hipMalloc((void**)&dev_indices, sizeof(int) * newN);
            checkCUDAError("Efficient scan: hipMalloc failed (dev_indices)");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            dim3 numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean <<<numBlocks, BLOCK_SIZE>>> (n, dev_bools, dev_idata);
            hipMemcpy(dev_indices, dev_bools, sizeof(int) * n, hipMemcpyDeviceToDevice);

            efficientScanUpDownSweep(n, newN, dev_indices);

            Common::kernScatter <<<numBlocks, BLOCK_SIZE>>> (n, dev_bools, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();
            int lastIdx;
            hipMemcpy(&lastIdx, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int lastBool;
            hipMemcpy(&lastBool, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            return lastIdx + lastBool;
        }
    }
}
