#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

static int blockSize = 64;

namespace StreamCompaction {
    namespace Efficient {
        
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        __global__ void kernelUpSweep(int n, int offset, int* i_odata) {
            int threadId = threadIdx.x + blockDim.x * blockIdx.x + 1;
            if (threadId > n)return;
            int id = offset * 2 * threadId - 1;
            i_odata[id] = i_odata[id] + i_odata[id - offset];
        }

        __global__ void kernelDownSweep(int n, int offset, int* i_odata) {
            int threadId = threadIdx.x + blockDim.x * blockIdx.x + 1;
            if (threadId > n)return;
            int id = offset * 2 * threadId - 1;
            //change 2
            int prevIdx = id - offset;
            int prevNum = i_odata[prevIdx];
            i_odata[prevIdx] = i_odata[id];
            i_odata[id] += prevNum;
        }

        void devScan(int* dev_data, int layerCnt, int blockSize) {
            int N = 1 << layerCnt;
            int offset = 1;
            int needN = N;
            for (int i = 0;i < layerCnt;++i) {
                needN /= 2;
                dim3 blockPerGrid((needN + blockSize - 1) / blockSize);
                kernelUpSweep << <blockPerGrid, blockSize >> > (needN, offset, dev_data);
                offset *= 2;
            }
            hipMemset(dev_data + offset - 1,0,sizeof(int));
            for (int i = 0;i < layerCnt;++i) {
                offset /= 2;
                dim3 blockPerGrid((needN + blockSize - 1) / blockSize);
                kernelDownSweep << <blockPerGrid, blockSize >> > (needN, offset, dev_data);
                needN *= 2;
            }
        }

        void scan(int n, int *odata, const int *idata) {
            
            // TODO
            int* dev_data;
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();

            devScan(dev_data, layerCnt, blockSize);

            timer().endGpuTimer();

            //exclusive scan
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_data);
            
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            
            // TODO
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;
            hipMalloc((void**)&dev_idata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_indices, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            
            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

            timer().startCpuTimer();

            StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize>>>(N, dev_bools, dev_idata);
            
            hipMemcpy(dev_indices, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
            devScan(dev_indices, layerCnt, blockSize);
            
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            
            timer().endCpuTimer();

            //read GPU
            int ans = 0;
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipMemcpy(&ans, dev_indices + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            int lastBool = 0;
            hipMemcpy(&lastBool, dev_bools + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            ans += lastBool;
            
            //free GPU
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_odata);
            hipFree(dev_indices);
            
            return ans;
        }

        __global__ void kernMapToBoolean(int n, int* bools, const int* idata, int mask, bool recordZero) {
            // TODO
            int id = blockDim.x * blockIdx.x + threadIdx.x;
            if (id >= n)return;
            bools[id] = (idata[id] & mask) == 0 ? recordZero : (!recordZero);
        }

        __global__ void kernSortScatter(int n, int* odata,
            const int* idata, const int* isZeroBools, 
            const int* indices_0,int zeroCnt) {
            // TODO
            int id = blockDim.x * blockIdx.x + threadIdx.x;
            if (id >= n)return;
            if (isZeroBools[id] == 1) {
                int idx = indices_0[id];
                odata[idx] = idata[id];
            }
            else {
                //ones before current id: id - indices_0[id], therefore we remove a scan 
                int idx = id - indices_0[id] + zeroCnt;
                odata[idx] = idata[id];
            }
        }

        void sort(int n, int* odata, const int* idata) {
            
            // TODO
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            //int* dev_indices_1;
            int* dev_indices_0;
            hipMalloc((void**)&dev_idata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            //hipMalloc((void**)&dev_indices_1, N * sizeof(int));
            //checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_indices_0, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(dev_idata + n, INT_MAX, (N - n)*sizeof(int));//to make non-power-of-two right

            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
            int mask = 1;

            timer().startGpuTimer();

            for (int i = 0;i < 32;++i) {
                //map 0
                kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (N, dev_bools, dev_idata,mask,true);
                hipMemcpy(dev_indices_0, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
                devScan(dev_indices_0, layerCnt, blockSize);
                int zeroCnt = 0;
                int lastBool = 0;
                hipMemcpy(&zeroCnt, dev_indices_0 + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastBool, dev_bools + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
                zeroCnt += lastBool;

                //map1
                //kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (N, dev_bools, dev_idata, mask, false);
                //hipMemcpy(dev_indices_1, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
                //devScan(dev_indices_1, layerCnt, blockSize);
                kernSortScatter << <fullBlocksPerGrid, blockSize >> > (N,
                    dev_odata, dev_idata, dev_bools, dev_indices_0,zeroCnt);
                mask <<= 1;
                std::swap(dev_odata, dev_idata);
            }

            timer().endGpuTimer();

            //read GPU
            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);

            //free GPU
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_odata);
            hipFree(dev_indices_0);
            
        }
    }
}
