#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        __global__ void kernelUpSweep(int n, int offset, int* i_odata) {
            int id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id >= n || (id + 1) % (offset * 2) != 0)return;
            i_odata[id] = i_odata[id] + i_odata[id - offset];
        }

        __global__ void kernelDownSweep(int n, int offset, int* i_odata) {
            int id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id >= n || (id+1)%(offset*2)!=0)return;
            //change 2
            int prevIdx = id - offset;
            int prevNum = i_odata[prevIdx];
            i_odata[prevIdx] = i_odata[id];
            i_odata[id] += prevNum;
        }

        void devScan(int* dev_data, int layerCnt, const dim3& fullBlocksPerGrid, int blockSize) {
            int N = 1 << layerCnt;
            int offset = 1;
            for (int i = 0;i < layerCnt;++i) {
                kernelUpSweep << <fullBlocksPerGrid, blockSize >> > (N, offset, dev_data);
                offset *= 2;
            }
            hipMemset(dev_data + offset - 1,0,sizeof(int));
            for (int i = 0;i < layerCnt;++i) {
                offset /= 2;
                kernelDownSweep << <fullBlocksPerGrid, blockSize >> > (N, offset, dev_data);
            }
        }

        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int* dev_data;
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            devScan(dev_data, layerCnt,fullBlocksPerGrid, blockSize);

            //exclusive scan
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_data);
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startCpuTimer();
            // TODO
            int layerCnt = ilog2ceil(n);
            int N = 1 << layerCnt;
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;
            hipMalloc((void**)&dev_idata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_indices, N * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            
            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
            StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid,blockSize>>>(N, dev_bools, dev_idata);
            
            hipMemcpy(dev_indices, dev_bools, N * sizeof(int), hipMemcpyDeviceToDevice);
            devScan(dev_indices, layerCnt, fullBlocksPerGrid, blockSize);
            
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (N, dev_odata, dev_idata, dev_bools, dev_indices);
            
            //read GPU
            int ans = 0;
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipMemcpy(&ans, dev_indices + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            int lastBool = 0;
            hipMemcpy(&lastBool, dev_bools + (N - 1), sizeof(int), hipMemcpyDeviceToHost);
            ans += lastBool;
            
            //free GPU
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_odata);
            hipFree(dev_indices);
            timer().endCpuTimer();
            return ans;
        }
    }
}
