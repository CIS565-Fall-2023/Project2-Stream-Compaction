#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define block_size 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernReduceIter(int n, int* idata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            // n here is not arr size but num threads needed on iter d
            if (index >= n) {
                return;
            }
            //from slides, k = index*offset
            int offset = 1 << (d + 1);
            idata[index * offset + (1 << (d + 1)) - 1] += idata[index * offset + (1 << d) - 1];
        }

        __global__ void kernDownSweepIter(int n, int* idata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            // n here is not arr size but num threads needed on iter d
            if (index >= n) {
                return;
            }
            //from slides, k = index*offset
            int offset = 1 << (d + 1);
            int tmp = idata[index * offset + (1 << d) - 1];
            idata[index * offset + (1 << d) - 1] = idata[index * offset + (1 << (d + 1)) - 1];
            idata[index * offset + (1 << (d + 1)) - 1] += tmp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            dim3 block_dim((n + block_size - 1) / block_size);
            int* dev_idata, *dev_odata;
            int log2n = ilog2ceil(n);
            int arr_size = 1 << log2n;
            hipMalloc((void**)&dev_idata, arr_size * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, arr_size * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            //memset pow of 2 arrays to 0 to preset padded vals
            hipMemset(dev_idata, 0, arr_size * sizeof(int));
            checkCUDAError("hipMemset dev_idata failed!");
            hipMemset(dev_odata, 0, arr_size * sizeof(int));
            checkCUDAError("hipMemset dev_odata failed!");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata into dev_idata failed!");

            timer().startGpuTimer();
            //up sweep in place
            for (int d = 0; d < log2n; d++) {
                kernReduceIter << <dim3((arr_size / (1 << (d + 1)) + block_size - 1) / block_size), block_size >> > (arr_size / (1 << (d + 1)), dev_idata, d);
            }
            //down sweep in place
            //set last elem to 0 through cuda(on dev)
            hipMemset(&dev_idata[arr_size-1], 0, sizeof(int));
            for (int d = log2n - 1; d >= 0; d--) {
                kernDownSweepIter << < dim3((arr_size / (1 << (d + 1)) + block_size - 1) / block_size), block_size >> > (arr_size / (1 << (d + 1)), dev_idata, d);
            }
            //swap so free logic below remains valid
            std::swap(dev_odata, dev_idata);
            timer().endGpuTimer();

            //only copy first n values(exclude tail padded 0s)
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata into odata failed!");
            hipFree(dev_odata);
            checkCUDAError("free dev_odata failed!");
            hipFree(dev_idata);
            checkCUDAError("free dev_idata failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            dim3 block_dim((n + block_size - 1) / block_size);
            int* dev_idata, *dev_odata, *dev_filter_map;
            int log2n = ilog2ceil(n);
            int arr_size = 1 << log2n;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            //padded for scan
            hipMalloc((void**)&dev_filter_map, arr_size * sizeof(int));
            checkCUDAError("hipMalloc dev_filter_map failed!");

            //memset pow of 2 arrays to 0 to preset padded vals
            hipMemset(dev_filter_map, 0, arr_size * sizeof(int));
            checkCUDAError("hipMemset dev_filter_map failed!");

            //copy input data to device
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata into dev_idata failed!");

            timer().startGpuTimer();
            //generate filter map for input in dev_filter_map
            StreamCompaction::Common::kernMapToBoolean << <block_dim, block_size >> > (n, dev_filter_map, dev_idata);
            //copy paste scan to avoid double timer start
            //in place on filter map since can use idata for bools

            //up sweep in place
            for (int d = 0; d < log2n; d++) {
                kernReduceIter << < dim3((arr_size / (1 << (d + 1)) + block_size - 1) / block_size), block_size >> > (arr_size / (1 << (d + 1)), dev_filter_map, d);
            }
            //down sweep in place
            //set last elem to 0 through cuda(on dev)
            hipMemset(&dev_filter_map[arr_size - 1], 0, sizeof(int));
            for (int d = log2n - 1; d >= 0; d--) {
                kernDownSweepIter << < dim3((arr_size / (1 << (d + 1)) + block_size - 1) / block_size), block_size >> > (arr_size / (1 << (d + 1)), dev_filter_map, d);
            }
            //dev_filter_map contains indices

            //final obj num
            int num_objs;
            hipMemcpy(&num_objs, &dev_filter_map[n - 1], sizeof(int), hipMemcpyDeviceToHost);
            //inc num_objs since exc scan
            if (idata[n - 1] != 0) num_objs++;

            //scatter on output array
            StreamCompaction::Common::kernScatter << <block_dim, block_size >> > (n, dev_odata, dev_idata, dev_idata, dev_filter_map);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * num_objs, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata into odata failed!");
            hipFree(dev_odata);
            checkCUDAError("free dev_odata failed!");
            hipFree(dev_idata);
            checkCUDAError("free dev_idata failed!");
            hipFree(dev_filter_map);
            checkCUDAError("free dev_filter_map failed!");
            return num_objs;
        }
    }
}
