#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#define BlockSize 256
namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpSweep(int n, int* odata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n || (index % (int)powf(2, d + 1) != 0)) {
                return;
            }

            odata[index + (int)powf(2, d + 1) - 1] += odata[index + (int)powf(2, d) - 1];
            
        }

        __global__ void kernZero(const int n, int* data) {
            data[n - 1] = 0;
        }

        __global__ void kernDownSweep(int n, int* odata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n || (index % (int)powf(2, d + 1) != 0)) {
                return;
            }

            int t = odata[index + (int)powf(2, d) - 1];
            odata[index + (int)powf(2, d) - 1] = odata[index + (int)powf(2, d + 1) - 1];
            odata[index + (int)powf(2, d + 1) - 1] += t;
        }

        __global__ void kernCompact(int n, int* idata, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            
            int k = 0;
            if (idata[index] != 0) {
                odata[index] = k;
                k++;
            }
        }

        __global__ void kernScatter(int n, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

         

        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        int nextPowerOf2(int n) {
            if (n <= 0) return 0;

            int power = 1;
            while (power < n) {
                power *= 2;
            }

            return power;
        }

        void scan(int n, int *odata, const int *idata) {
            int* device_A;

            int paddedSize = nextPowerOf2(n);
            hipMalloc((void**)&device_A, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_A failed!");


            hipMemset(device_A + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_A hipMemset failed!");

            

            hipMemcpy(device_A, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy hipMemcpyHostToDevice device_A to idata failed!");

            dim3 blocksPerGrid((paddedSize + BlockSize - 1) / BlockSize);

            timer().startGpuTimer();
            
            for (int d = 0; d <= ilog2ceil(paddedSize) - 1; d++) { //Upsweep
                kernUpSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }

            kernZero << <1, 1 >> > (paddedSize, device_A);

            for (int d = ilog2ceil(paddedSize) - 1; d >= 0; d--) { //Downsweep
                kernDownSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }


            timer().endGpuTimer();

            
            hipMemcpy(odata, device_A, n * sizeof(int), hipMemcpyDeviceToHost);
            


            checkCUDAError("hipMemcpy hipMemcpyDeviceToHost odata to device_A failed!");

            hipFree(device_A);
            checkCUDAError("hipFree device_A failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* device_A;
            int* device_Binary;

            int paddedSize = nextPowerOf2(n);
            hipMalloc((void**)&device_A, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_A failed!");
            hipMalloc((void**)&device_Binary, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_Binary failed!");

            hipMemcpy(device_A, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy hipMemcpyHostToDevice device_A to idata failed!");


            hipMemset(device_A + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_A hipMemset failed!");
            hipMemset(device_Binary + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_Binary hipMemset failed!");

            dim3 blocksPerGrid((paddedSize + BlockSize - 1) / BlockSize);

            timer().startGpuTimer();

            for (int d = 0; d <= ilog2ceil(paddedSize) - 1; d++) { //Upsweep
                kernUpSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }

            kernZero << <1, 1 >> > (paddedSize, device_A);

            for (int d = ilog2ceil(paddedSize) - 1; d >= 0; d--) { //Downsweep
                kernDownSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }
            kernCompact << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, device_Binary);

            timer().endGpuTimer();

            hipMemcpy(odata, device_Binary, n * sizeof(int), hipMemcpyDeviceToHost);

            for (int i = 0; i < n; i++) {
                printf("%d ", odata[i]);
            }


            checkCUDAError("hipMemcpy hipMemcpyDeviceToHost odata to device_A failed!");

            hipFree(device_A);
            checkCUDAError("hipFree device_A failed!");
            return -1;
        }
    }
}
