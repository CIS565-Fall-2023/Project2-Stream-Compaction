#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* data) 
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;

            int powerd = 1 << d;
            int powerdp1 = 1 << (d + 1);

            if (k >= n || k % powerdp1) return;

            data[k + powerdp1 - 1] = data[k + powerd - 1] + data[k + powerdp1 - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* data) 
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;           

            int powerd = 1 << d;
            int powerdp1 = 1 << (d + 1);

            if (k >= n || k % powerdp1 || k + powerdp1 - 1 >= n) return;

            int t = data[k + powerd - 1];
            data[k + powerd - 1] = data[k + powerdp1 - 1];
            data[k + powerdp1 - 1] = t + data[k + powerdp1 - 1];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
        {
            int N = 1 << ilog2ceil(n);

            int* dev_data;

            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMemset(dev_data, 0, N * sizeof(int));
            checkCUDAErrorFn("hipMemset dev_data to 0 failed!");
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy idata to dev_data failed!");

            dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
            for (int d = 0; d < ilog2ceil(N); d++)
            {
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }

            hipMemset(dev_data + N - 1, 0, sizeof(int));
            for (int d = ilog2ceil(N) - 1; d >= 0; d--) 
            {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy dev_data to odata failed!");

            hipFree(dev_data);
            checkCUDAErrorFn("hipFree dev_data failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
        {
            int N = 1 << ilog2ceil(n);

            int* dev_bools;
            int* dev_data;
            int* dev_idata;
            int* dev_odata;

            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_data, N * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_odata failed!");

            hipMemset(dev_data, 0, N * sizeof(int));
            checkCUDAErrorFn("hipMemset dev_data failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("cudaMempcy idata to dev_idata failed!");

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
            StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_bools, dev_idata);

            hipMemcpy(dev_data, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAErrorFn("cudaMempcy dev_bools to dev_data failed!");

            for (int d = 0; d < ilog2ceil(N); d++)
            {
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }

            hipMemset(dev_data + N - 1, 0, sizeof(int));
            for (int d = ilog2ceil(N) - 1; d >= 0; d--)
            {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(N, d, dev_data);
            }

            StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_data);

            timer().endGpuTimer();

            int count = 0;
            hipMemcpy(&count, dev_data + N - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("cudaMempcy count failed!");
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("cudaMempcy dev_odata to odata failed!");

            hipFree(dev_bools);
            checkCUDAErrorFn("hipFree dev_bools failed!");
            hipFree(dev_data);
            checkCUDAErrorFn("hipFree dev_data failed!");
            hipFree(dev_idata);
            checkCUDAErrorFn("hipFree dev_idata failed!");
            hipFree(dev_odata);
            checkCUDAErrorFn("hipFree dev_odata failed!");

            return count;
        }
    }
}
