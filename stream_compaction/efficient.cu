#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int offset, int* x) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
				return;
			}

            int k = index * offset;
            x[k + offset - 1] += x[k + offset / 2 - 1];
        }

        __global__ void kernDownSweep(int n, int offset, int* x) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int k = index * offset;
            int t = x[k + offset / 2 - 1];
            x[k + offset / 2 - 1] = x[k + offset - 1];
            x[k + offset - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // memory operation
            int max_d = ilog2ceil(n);
            int next_power_of_two = pow(2, max_d);

            int* x;
            hipMalloc((void**)&x, next_power_of_two * sizeof(int));
            hipMemcpy(x, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            // TODO
            int blockSize = 64;

            // up-sweep
            for (int d = 0; d < max_d; ++d) {
                int step = pow(2, d + 1);
                int threadCount = next_power_of_two / step;
                dim3 fullBlocksPerGrid((threadCount + blockSize - 1) / blockSize);

				kernUpSweep<<<fullBlocksPerGrid, blockSize >>>(threadCount, step, x);
			}

            // down-sweep
            hipMemset(x + next_power_of_two - 1, 0, sizeof(int));
            for (int d = max_d - 1; d >= 0; --d) {
				int step = pow(2, d + 1);
                int threadCount = next_power_of_two / step;
				dim3 fullBlocksPerGrid((threadCount + blockSize - 1) / blockSize);

                kernDownSweep<<<fullBlocksPerGrid, blockSize >>>(threadCount, step, x);
            }

            timer().endGpuTimer();

            hipMemcpy(odata, x, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(x);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* bools, *scanArr, *out, *in;
            hipMalloc((void**)&bools, n * sizeof(int));
            hipMalloc((void**)&scanArr, n * sizeof(int));
            hipMalloc((void**)&out, n * sizeof(int));
            hipMalloc((void**)&in, n * sizeof(int));

            hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            //timer().startGpuTimer();
            
            // TODO
            int blockSize = 64;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // Step 1: Compute temporary array of 0s and 1s
            StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize >>>(n, bools, in);

            // Step2: Run exclusive scan on tempArr
            scan(n, scanArr, bools);

            // Step 3: Scatter
            StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, blockSize >>>(n, out, in, bools, scanArr);

            //timer().endGpuTimer();

            int count = 0, lastScan = 0;
            hipMemcpy(&count, scanArr + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastScan, bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(bools);
            hipFree(scanArr);
            hipFree(out);
            hipFree(in);

            return count + lastScan;
        }
    }
}
