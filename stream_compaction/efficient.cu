#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#define BlockSize 256
namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpSweep(int n, int* odata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n || (index % (int)powf(2, d + 1) != 0)) {
                return;
            }

            odata[index + (int)powf(2, d + 1) - 1] += odata[index + (int)powf(2, d) - 1];
            
        }
        __global__ void kernUpCopy(int n, int* idata, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (odata[index] != idata[index]) {
                odata[index] = idata[index];
            }

        }

        __global__ void kernZero(const int n, int* data) {
            data[n - 1] = 0;
        }

        __global__ void kernDownSweep(int n, int* odata, int d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n || (index % (int)powf(2, d + 1) != 0)) {
                return;
            }

            int t = odata[index + (int)powf(2, d) - 1];
            odata[index + (int)powf(2, d) - 1] = odata[index + (int)powf(2, d + 1) - 1];
            odata[index + (int)powf(2, d + 1) - 1] += t;
        }




        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        int nextPowerOf2(int n) {
            if (n <= 0) return 0;

            int power = 1;
            while (power < n) {
                power *= 2;
            }

            return power;
        }

        void scan(int n, int *odata, const int *idata) {
            int* device_A;

            int paddedSize = nextPowerOf2(n);
            hipMalloc((void**)&device_A, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_A failed!");


            hipMemset(device_A + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_A hipMemset failed!");

            

            hipMemcpy(device_A, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy hipMemcpyHostToDevice device_A to idata failed!");

            dim3 blocksPerGrid((paddedSize + BlockSize - 1) / BlockSize);

            timer().startGpuTimer();
            
            for (int d = 0; d <= ilog2ceil(paddedSize) - 1; d++) { //Upsweep
                kernUpSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }

            kernZero << <1, 1 >> > (paddedSize, device_A);

            for (int d = ilog2ceil(paddedSize) - 1; d >= 0; d--) { //Downsweep
                kernDownSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_A, d);
            }


            timer().endGpuTimer();

            
            hipMemcpy(odata, device_A, n * sizeof(int), hipMemcpyDeviceToHost);
            


            checkCUDAError("hipMemcpy hipMemcpyDeviceToHost odata to device_A failed!");

            hipFree(device_A);
            checkCUDAError("hipFree device_A failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* device_idata, * device_odata, * device_bool, * device_scan;

            int paddedSize = nextPowerOf2(n);
            hipMalloc((void**)&device_idata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_idata failed!");
            hipMalloc((void**)&device_odata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_odata failed!");
            hipMalloc((void**)&device_bool, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_bool failed!");
            hipMalloc((void**)&device_scan, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc device_scan failed!");

            hipMemcpy(device_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy hipMemcpyHostToDevice device_idata to idata failed!");


            hipMemset(device_idata + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_idata hipMemset failed!");
            hipMemset(device_scan + n, 0, (paddedSize - n) * sizeof(int));
            checkCUDAError("device_scan hipMemset failed!");

            dim3 blocksPerGrid((paddedSize + BlockSize - 1) / BlockSize);

            timer().startGpuTimer();
            StreamCompaction::Common::kernMapToBoolean << <blocksPerGrid, BlockSize >> > (n, device_bool, device_idata);
            kernUpCopy << <blocksPerGrid, BlockSize >> > (n, device_bool, device_scan);


            for (int d = 0; d <= ilog2ceil(paddedSize) - 1; d++) { //Upsweep
                kernUpSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_scan, d);
            }

            kernZero << <1, 1 >> > (paddedSize, device_scan);

            for (int d = ilog2ceil(paddedSize) - 1; d >= 0; d--) { //Downsweep
                kernDownSweep << <blocksPerGrid, BlockSize >> > (paddedSize, device_scan, d);
            }

            StreamCompaction::Common::kernScatter << <blocksPerGrid, BlockSize >> > (paddedSize, device_odata, device_idata, device_bool, device_scan);
            timer().endGpuTimer();

            hipMemcpy(odata, device_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            int finalSize;
            hipMemcpy(&finalSize, device_scan + paddedSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy hipMemcpyDeviceToHost odata to device_idata failed!");

            hipFree(device_idata);
            checkCUDAError("hipFree device_idata failed!");
            hipFree(device_odata);
            checkCUDAError("hipFree device_odata failed!");
            hipFree(device_bool);
            checkCUDAError("hipFree device_bool failed!");
            hipFree(device_scan);
            checkCUDAError("hipFree device_scan failed!");
            return finalSize;
        }
    }
}
