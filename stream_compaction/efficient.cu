#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
#include "common.cu"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void initData(int n, int max, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < (max - n)) {
                data[n + index] = 0;
            }
        }

        __global__ void changeNum(int i, int newNum, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index == 0) {
                data[i] = newNum;
            }
        }
        
        __global__ void upSweep(int N, int offsetBase, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            int offset = 1 << offsetBase;
            if (index * offset * 2 - 1 < N) {
                data[index * offset * 2 - 1] += data[index * offset * 2 - offset - 1];
            }
        }

        __global__ void downSweep(int N, int offsetBase, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            int offset = 1 << offsetBase;
            if (index * offset * 2 - 1 < N) {
                int t = data[index * offset * 2 - offset - 1];
                data[index * offset * 2 - offset - 1] = data[index * offset * 2 - 1];
                data[index * offset * 2 - 1] += t;
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            const int blockSize = 32;
            const int max = 1 << (ilog2ceil(n - 1) + 1);
            // std::cout << "n = " << n << ", max = " << max << std::endl;

            int* dev_data;
            hipMalloc((void**)&dev_data, max * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata -> dev_data failed!");

            timer().startGpuTimer();
            //if (max > n) {
            //    dim3 initBlockNum((max - n + blockSize - 1) / blockSize);
            //    initData << <initBlockNum, blockSize >> > (n, max, dev_data);
            //}

            // up sweep
            int addTimes = max / 2;
            for (int i = 0; i < ilog2ceil(max); i++) {
                dim3 fullBlocksPerGrid((addTimes + blockSize) / blockSize);
                upSweep << <fullBlocksPerGrid, blockSize >> > (max, i, dev_data);
                addTimes /= 2;
            }

            // down sweep
            int swapTime = 1;
            changeNum << <1, 1 >> > (max - 1, 0, dev_data);

            for (int i = ilog2ceil(max) - 1; i >= 0; i--) {
                dim3 fullBlocksPerGrid((swapTime + blockSize) / blockSize);
                downSweep << <fullBlocksPerGrid, blockSize >> > (max, i, dev_data);
                swapTime *= 2;
            }

            //hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            //checkCUDAError("hipMemcpy dev_data -> odata for cout failed!");
            //for (int i = 0; i < n; i++) { std::cout << odata[i] << ", "; }
            //std::cout << std::endl << std::endl << std::endl << std::endl;

            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_data -> odata failed!");

            hipFree(dev_data);
            checkCUDAError("hipFree failed!");
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

            const int blockSize = 32;
            int* dev_idata;
            int* dev_odata;
            int* dev_bool;
            int* dev_sum;
            const int max = 1 << (ilog2ceil(n - 1) + 1);
            hipMalloc((void**)&dev_idata, max * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_bool, max * sizeof(int));
            checkCUDAError("hipMalloc dev_bool failed!");
            hipMalloc((void**)&dev_sum, max * sizeof(int));
            checkCUDAError("hipMalloc dev_sum failed!");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata -> dev_data failed!");

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            // checkCUDAError("hipMemcpy idata -> dev_data failed!");

            timer().startGpuTimer();
            
            Common::kernMapToBoolean << < fullBlocksPerGrid, blockSize >> > (max, dev_bool, dev_idata);
            hipMemcpy(dev_sum, dev_bool, sizeof(int) * max, hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy idata -> dev_data failed!");

            // up sweep
            
            int addTimes = max / 2;
            for (int i = 0; i < ilog2ceil(max); i++) {
                dim3 upFullBlocksPerGrid((addTimes + blockSize) / blockSize);
                upSweep << <upFullBlocksPerGrid, blockSize >> > (max, i, dev_sum);
                addTimes /= 2;
            }

            // down sweep
            int swapTime = 1;
            changeNum << <1, 1 >> > (max - 1, 0, dev_sum);

            for (int i = ilog2ceil(max) - 1; i >= 0; i--) {
                dim3 downFullBlocksPerGrid((swapTime + blockSize) / blockSize);
                downSweep << <downFullBlocksPerGrid, blockSize >> > (max, i, dev_sum);
                swapTime *= 2;
            }

            Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bool, dev_sum);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata -> odata failed!");

            int res_n = -1;
            int last_bool = 0;
            hipMemcpy(&res_n, dev_sum + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata_last -> res_n failed!");
            hipMemcpy(&last_bool, dev_bool + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_bool_last -> last_bool failed!");
            res_n += last_bool;

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bool);
            hipFree(dev_sum);
            checkCUDAError("hipFree failed!");

            return res_n;
        }
    }
}
