#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernelScan(int n, int* odata, const int* idata, int offset)
        {
            int index = (blockDim.x * blockIdx.x) + threadIdx.x;
            if (index >= n)
            {
                return;     // invalid index
            }

            if (index >= offset)
            {
                odata[index] = idata[index - offset] + idata[index];
            }
            else
            {
                odata[index] = idata[index];
            }
        }

        __global__ void kernelShiftRight(int n, int* odata, int* idata)
        {
            int index = (blockDim.x * blockIdx.x) + threadIdx.x;
            if (index >= n)
            {
                return;     // invalid index
            }

            if (index == 0)
            {
                odata[index] = 0;
            }
            else
            {
                odata[index] = idata[index - 1];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_idata failed");
            hipMalloc((void**)&dev_odata, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_odata failed");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed");
            hipMemcpy(dev_odata, odata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_odata failed");

            int totalBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            timer().startGpuTimer();
            int max = ilog2ceil(n);
            for (int d = 1; d <= max; d++)
            {
                kernelScan<<<totalBlocks, BLOCK_SIZE>>>(n, dev_odata, dev_idata, 1 << d - 1);
                std::swap(dev_odata, dev_idata);
            }
            kernelShiftRight<<<totalBlocks, BLOCK_SIZE>>>(n, dev_odata, dev_idata);
            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata to odata failed");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed");
            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed");
        }
    }
}
