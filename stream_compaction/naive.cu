#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define logBlockSize 6

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernAdd(int d, int* odata, const int* idata) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;
            int space = 1 << (d - 1);
            if (k >= space) {
                odata[k] = idata[k] + idata[k - space];
            }
            else {
                odata[k] = idata[k];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            const int blockSize = 1 << logBlockSize;

            int* dev_odata;
            int* dev_idata;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, n * sizeof(int));

            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            for (int i = 1; i <= ilog2ceil(n); i++) {
                kernAdd << <fullBlocksPerGrid, blockSize >> > (i, dev_odata, dev_idata);
                std::swap(dev_odata, dev_idata);
            }

            timer().endGpuTimer();

            hipMemcpy(odata + 1, dev_idata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_idata);
            
        }
    }
}
