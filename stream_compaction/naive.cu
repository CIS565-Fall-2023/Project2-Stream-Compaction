#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define BlockSize 256
namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        __global__ void KernNaiveScan(int n, int d, int* odata, int* idata)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index > n) return;
            int path = 1 << (d - 1);
            if (index >= path)
            {
                odata[index] = idata[index - path] + idata[index];
            }
            else
            {
                odata[index] = idata[index];
            }
        }

        void scan(int n, int* odata, const int* idata) {
            dim3 BlockDim((n + BlockSize - 1) / BlockSize);

            int* dev_odata;
            int* dev_odata2;
            hipMalloc((void**)&dev_odata2, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata1 failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata2 failed!");

            hipMemcpy(dev_odata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_odata2, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            int depth = ilog2ceil(n);

            timer().startGpuTimer();

            for (int d = 1; d <= depth; ++d) {
                KernNaiveScan <<<BlockDim, BlockSize >>> (n, d, dev_odata, dev_odata2);
                hipDeviceSynchronize();
                int* temp = dev_odata2;
                dev_odata2 = dev_odata;
                dev_odata = temp;
            }

            timer().endGpuTimer();
            hipMemcpy(odata + 1, dev_odata2, (n - 1) *sizeof(int), hipMemcpyDeviceToHost);
            odata[0] = 0;
            hipFree(dev_odata);
            hipFree(dev_odata2);
        }

    }
}
