#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

__global__ void kernNaiveScan(int offset, int n, int* odata, const int* idata) {
    int k = threadIdx.x + (blockIdx.x * blockDim.x);
    if (k >= n) {
        return;
    }
    odata[k] = idata[k];
    if (k >= offset) {
        odata[k] += idata[k - offset];
    }
}

__global__ void kernRightShift(int n, int* data) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= n - 1) {
        return;
    }
    int readVal = data[index];
    __syncthreads();
    data[index + 1] = readVal;
}

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            int* dev_in;
            int* dev_out;           
            int noOfIters = ilog2ceil(n);
            dim3 fullBlocksPerGrid((n + BLOCKSIZE - 1) / BLOCKSIZE);

            hipMalloc((void**)&dev_in, n * sizeof(int));
            checkCUDAError("hipMalloc dev_in failed!");

            hipMalloc((void**)&dev_out, n * sizeof(int));
            checkCUDAError("hipMalloc dev_out failed!");            
            
            hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            timer().startGpuTimer();
            for (int d = 1; d <= noOfIters; d++) {
                int offset = 1 << (d - 1);
                kernNaiveScan << <fullBlocksPerGrid, BLOCKSIZE >> > (offset, n, dev_out, dev_in);
                std::swap(dev_in, dev_out);
            }
            std::swap(dev_in, dev_out);
            kernRightShift << <fullBlocksPerGrid, BLOCKSIZE >> > (n, dev_out);
            hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
            odata[0] = 0;            
            timer().endGpuTimer();
            hipFree(dev_in);
            hipFree(dev_out);
        }
    }
}
