#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // TODO: __global__

        __global__ void kernNaiveScan(int n, int offset, int* x, const int* last) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
				return;
			}

            x[index] = last[index];
            if (index >= offset) {
                x[index] += last[index - offset];
            }
        }

        __global__ void kernShift(int n, int* x, const int* last) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            x[index] = index == 0 ? 0 : last[index - 1];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // memory operation
            int *x, *last;
            hipMalloc((void**)&x, n * sizeof(int));
            hipMalloc((void**)&last, n * sizeof(int));
            hipMemcpy(last, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            int blockSize = 64;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int max_d = ilog2ceil(n);
            // TODO
            for (int d = 1; d <= max_d; ++d) {
                kernNaiveScan<<<fullBlocksPerGrid, blockSize >>>(n, pow(2, d - 1), x, last);
				std::swap(x, last);
			}

            kernShift<<<fullBlocksPerGrid, blockSize >>>(n, x, last);

            timer().endGpuTimer();

            hipMemcpy(odata, x, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(x);
            hipFree(last);
        }
    }
}
