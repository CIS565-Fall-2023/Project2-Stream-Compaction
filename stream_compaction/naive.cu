#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#define BlockSize 256

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernScan(int n, int* odata, const int* idata, int d) {
             int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            if (odata[index] != idata[index]) {
                odata[index] = idata[index];
            }
            
            if (index >= (int)powf(2, d - 1)) {
                odata[index] = idata[index - (int)powf(2, d - 1)] + idata[index];
            }
        }

        __global__ void kernelIncToExc(const int n, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }

            
            if (index == 0) {
                odata[0] = 0;
            }
            else {
                odata[index] = idata[index - 1];
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* device_A;
            int* device_B;

  
            hipMalloc((void**)&device_A, n * sizeof(int));
            checkCUDAError("hipMalloc device_A failed!");
            hipMalloc((void**)&device_B, n * sizeof(int));
            checkCUDAError("hipMalloc device_B failed!");
            
            hipMemcpy(device_A, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy hipMemcpyHostToDevice device_A to idata failed!");

            dim3 blocksPerGrid((n + BlockSize - 1) / BlockSize);

            timer().startGpuTimer();
            // TODO
            int* temp;
            for (int d = 1; d <= ilog2ceil(n); d++) {
                kernScan << <blocksPerGrid, BlockSize >> > (n, device_B, device_A, d);
                temp = device_A;
                device_A = device_B;
                device_B = temp;
            }

            kernelIncToExc << <blocksPerGrid, BlockSize >> > (n, device_B, device_A);
            

            timer().endGpuTimer();

            hipMemcpy(odata, device_B, n * sizeof(int), hipMemcpyDeviceToHost);
            
            checkCUDAError("hipMemcpy hipMemcpyDeviceToHost odata to device_B failed!");

            hipFree(device_A);
            checkCUDAError("hipFree device_A failed!");
            hipFree(device_B);
            checkCUDAError("hipFree device_B failed!");

        }
    }
}
