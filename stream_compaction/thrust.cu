#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            
            hipMalloc(&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc(&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            thrust::device_ptr<int> dev_in_ptr(dev_idata);
            thrust::device_ptr<int> dev_out_ptr(dev_odata);
            thrust::device_vector<int> dv_in(dev_in_ptr, dev_in_ptr +n);
            thrust::device_vector<int> dv_out(dev_out_ptr, dev_out_ptr +n);
            
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            timer().endGpuTimer();
            hipMemcpy(odata, dv_out.data().get(), n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_odata);
        }
    }
}
