#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            int* dev_odata;
            int* dev_idata;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, n * sizeof(int));

            hipMemcpy(dev_odata, odata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            thrust::device_ptr<int> dev_thrust_odata(dev_odata);
            thrust::device_ptr<int> dev_thrust_idata(dev_idata);

            timer().startGpuTimer();

            thrust::exclusive_scan(dev_thrust_idata, dev_thrust_idata + n, dev_thrust_odata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            
        }
    }
}
