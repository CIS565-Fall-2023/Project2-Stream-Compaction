#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

            int* gpu_odata;
            int* gpu_idata;

            hipMalloc((void**)&gpu_odata, n * sizeof(int));
            hipMalloc((void**)&gpu_idata, n * sizeof(int));
            hipMemcpy(gpu_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            thrust::device_ptr<int>thrust_dv_in(gpu_idata);
            thrust::device_ptr<int>thrust_dv_out(gpu_odata);

            thrust::exclusive_scan(thrust_dv_in, thrust_dv_in+n, thrust_dv_out);




            hipMemcpy(odata, gpu_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(gpu_odata);
            hipFree(gpu_idata);


            timer().endGpuTimer();
        }
    }
}
