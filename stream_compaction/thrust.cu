#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            

            thrust::device_vector<int> dv_in(idata, idata + n);
            thrust::device_vector<int> dv_out(n);
            timer().startGpuTimer();
            thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            timer().endGpuTimer();
            thrust::copy(dv_out.begin()+1, dv_out.end(), odata);
            odata[n-1] = odata[n-2] + idata[n-1];
        }
    }
}
