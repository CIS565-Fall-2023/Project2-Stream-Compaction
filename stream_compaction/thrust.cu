#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <vector>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::device_vector<int> in_vec(idata, idata + n);
            thrust::device_vector<int> out_vec(n);
            hipDeviceSynchronize();
            timer().startGpuTimer();
            thrust::exclusive_scan(in_vec.begin(), in_vec.end(), out_vec.begin());
            timer().endGpuTimer();
            thrust::copy(out_vec.begin(), out_vec.end(), odata);
        }
    }
}
