#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
	namespace RadixSort {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		void sort(int n, int *odata, const int *idata) {
			timer().startGpuTimer();

			timer().endGpuTimer();
		}
	}
}
