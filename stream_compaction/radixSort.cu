#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "efficient.h"
#include "radixSort.h"

namespace StreamCompaction {
    namespace RadixSort {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernMapToBoolean(int n, int* odata, const int* idata, int mask) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            odata[index] = (idata[index] & mask) == 0;
        }

        __global__ void kernScatter(int n, int* odata, const int* idata, const int* falses, int mask, int totalFalses) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            int falsesIndex = falses[index];
            if ((idata[index] & mask) == 0) {
                odata[falsesIndex] = idata[index];
            }
            else {
                odata[index - falsesIndex + totalFalses] = idata[index];
            }
        }

        void sort(int n, int* odata, const int* idata, int numBits) {
            int extended_n = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid = BLOCKSPERGRID(n, blockSize);
            int* dev_idata;
            int* dev_odata;
            int* dev_falses;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_falses, extended_n * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            unsigned maxMask = 1 << numBits;
            for (unsigned mask = 1; mask < maxMask; mask <<= 1)
            {
                kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_falses, dev_idata, mask);
                StreamCompaction::Efficient::scanInplace(extended_n, dev_falses);
                int totalFalses = 0, tmp_back = 0;
                hipMemcpy(&totalFalses, dev_falses + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&tmp_back, dev_idata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                totalFalses += (tmp_back & mask) == 0;
                kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_falses, mask, totalFalses);
                std::swap(dev_idata, dev_odata);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_falses);
        }

        void sortShared(int n, int* odata, const int* idata, int numBits) {
            int extended_n = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid = BLOCKSPERGRID(n, blockSize);
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            Common::devDataBuffer buffer(extended_n, blockSize);
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            unsigned int maxMask = 1 << numBits;
            for (unsigned mask = 1; mask < maxMask; mask <<= 1)
            {
                kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, buffer.data(), dev_idata, mask);
                StreamCompaction::Efficient::scanSharedInplace(extended_n, buffer);
                int totalFalses = 0, tmp_back = 0;
                hipMemcpy(&totalFalses, buffer.data() + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&tmp_back, dev_idata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                totalFalses += (tmp_back & mask) == 0;
                kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, buffer.data(), mask, totalFalses);
                std::swap(dev_idata, dev_odata);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_odata);
        }

    }
}
